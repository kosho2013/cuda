#include <hip/hip_runtime.h>
#include <iostream>

__global__ void shiftLeftKernel(int *arr, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n - 1) {
        arr[idx] = arr[idx + 1];
    } else if (idx == n - 1) {
        // Handle the last element separately
        arr[idx] = arr[0];
    }
}

void shiftLeft(int *arr, int n) {
    int *d_arr;
    size_t size = n * sizeof(int);

    hipMalloc(&d_arr, size);
    hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    shiftLeftKernel<<<blocksPerGrid, threadsPerBlock>>>(d_arr, n);

    hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main() {
    const int n = 10;
    int arr[n] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};

    shiftLeft(arr, n);

    std::cout << "Shifted array: ";
    for (int i = 0; i < n; i++) {
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}
