#include <iostream>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>

using namespace nvcuda;

const int M = 16;  // Tile dimensions for Tensor Cores (16x16x16)
const int N = 16;
const int K = 16;
const int TILE_DIM = 1024; // GEMM size

__global__ void tensorCoreGemm1024x1024x1024(half *a, half *b, float *c, int TILE_DIM) {
    __shared__ half shared_a[M * K];
    __shared__ half shared_b[K * N];

    wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, M, N, K, float> c_frag;

    int tileRow = blockIdx.y * blockDim.y + threadIdx.y;
    int tileCol = blockIdx.x * blockDim.x + threadIdx.x;

    wmma::fill_fragment(c_frag, 0.0f);

    for (int tileIdx = 0; tileIdx < TILE_DIM / K; ++tileIdx) {
        shared_a[threadIdx.y * K + threadIdx.x] = a[(tileRow * TILE_DIM) + (tileIdx * K + threadIdx.x)];
        shared_b[threadIdx.y * K + threadIdx.x] = b[(tileIdx * K + threadIdx.y) * TILE_DIM + tileCol];
        __syncthreads();

        wmma::load_matrix_sync(a_frag, &shared_a[0], K);
        wmma::load_matrix_sync(b_frag, &shared_b[0], N);

        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        __syncthreads();
    }

    wmma::store_matrix_sync(&c[tileRow * TILE_DIM + tileCol], c_frag, TILE_DIM, wmma::mem_row_major);
}

int main() {
    const int MATRIX_SIZE_A = TILE_DIM * TILE_DIM * sizeof(half);
    const int MATRIX_SIZE_B = TILE_DIM * TILE_DIM * sizeof(half);
    const int MATRIX_SIZE_C = TILE_DIM * TILE_DIM * sizeof(float);

    half *a, *b;
    float *c;

    hipMalloc(&a, MATRIX_SIZE_A);
    hipMalloc(&b, MATRIX_SIZE_B);
    hipMalloc(&c, MATRIX_SIZE_C);

    // Host-side arrays for initialization and verification
    half *host_a = new half[TILE_DIM * TILE_DIM];
    half *host_b = new half[TILE_DIM * TILE_DIM];
    float *host_c = new float[TILE_DIM * TILE_DIM];

    for (int i = 0; i < TILE_DIM * TILE_DIM; ++i) {
        host_a[i] = __float2half(1.0f);  // Fill with 1s
        host_b[i] = __float2half(1.0f);  // Fill with 1s
    }

    // Copy initialized arrays to device
    hipMemcpy(a, host_a, MATRIX_SIZE_A, hipMemcpyHostToDevice);
    hipMemcpy(b, host_b, MATRIX_SIZE_B, hipMemcpyHostToDevice);

    dim3 grid(TILE_DIM / M, TILE_DIM / N);
    dim3 block(M, N);

    // Initialize CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start);

    // Launch the GEMM kernel
    tensorCoreGemm1024x1024x1024<<<grid, block>>>(a, b, c, TILE_DIM);

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed time in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Convert time to nanoseconds
    float nanoseconds = milliseconds * 1e6;

    // Calculate FLOPS
    long long int num_operations = 2LL * TILE_DIM * TILE_DIM * TILE_DIM;
    float tflops = (num_operations / (nanoseconds / 1e9)) / 1e12; // TFLOPS

    std::cout << "Time: " << nanoseconds << " ns" << std::endl;
    std::cout << "Achieved TFLOPS: " << tflops << std::endl;

    // Copy result matrix C back to host for verification
    hipMemcpy(host_c, c, MATRIX_SIZE_C, hipMemcpyDeviceToHost);

    // Verify correctness by checking if each element in C is 1024
    bool correct = true;
    float expected_value = 1024.0f;
    for (int i = 0; i < TILE_DIM * TILE_DIM; ++i) {
        if (abs(host_c[i] - expected_value) > 1e-3) {  // Allow small floating-point tolerance
            correct = false;
            std::cout << "Mismatch at index " << i << ": " << host_c[i] << " != " << expected_value << std::endl;
            break;
        }
    }
    if (correct) {
        std::cout << "Matrix multiplication result is correct!" << std::endl;
    } else {
        std::cout << "Matrix multiplication result is incorrect." << std::endl;
    }

    // Cleanup
    delete[] host_a;
    delete[] host_b;
    delete[] host_c;
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
