#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

__global__ void shift(int *a, int *b, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        b[i] = a[i];
    }
}


int main() {
    int n = 10;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = n * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<1, n>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
        printf("%d + %d = %d\n", a[i], b[i], c[i]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);






    int n = 10;
    int *a, *b;
    int *d_a, *d_b;
    int size = n * sizeof(int);


    a = (int *)malloc(size);
    b = (int *)malloc(size);

    for (int i = 0; i < n; i++)
    {
        a[i] = i;
    }

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    shift<<<1, n>>>(d_a, d_b, n);

    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
    {
        printf(a[i], b[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    free(a);
    free(b);


    return 0;
}